#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cudnn.h>
int main() {
    int cuda_version = 0;
    
    // Get the version of the installed CUDA runtime
    hipRuntimeGetVersion(&cuda_version);
    
    // Print the version in major.minor format
    printf("CUDA Version: %d.%d\n", cuda_version / 1000, (cuda_version % 1000) / 10);
   
    // cuDNN version
    printf("cuDNN Version: %d\n", CUDNN_VERSION);  // CUDNN_VERSION is a predefined macro

    return 0;
}

